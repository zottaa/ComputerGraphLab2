#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>


//function declarations
__global__ void editRows(double *data, int matrixOrder, int rowNumber);

__host__ double gaussDeterminant (double* data, double* dev_data, int matrixOrder);

int main() {
    FILE *f1, *f2;
    //main cycle
    for (int n = 100; n < 1000; n += 20) {
        //matrix creation
        double *data = (double *) malloc(sizeof(double) * (n * n));

        srand(time(0));
        for (int i = 0; i < n * n; i++) {
            data[i] = (double) (rand() % 101);
        }

        double determinant = 1;
        //write matrix order to file
        f1 = fopen("Matrix Orders.txt", "a");
        printf("matrix order = %d\n", n);
        fprintf(f1, "%d\n", n);
        fclose(f1);

        double *dev_data;
        //init events to get time of calculation
        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, 0);

        //allocate memory on device
        hipMalloc((void **) &dev_data, sizeof(double) * (n * n));
        hipMemcpy(dev_data, data, sizeof(double) * n * n, hipMemcpyHostToDevice);

        float timeToCalculate = 0.0;

        determinant = gaussDeterminant(data, dev_data, n);

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&timeToCalculate, begin, end);

        //write time to calculate in file
        f2 = fopen("Time in milli seconds GPU.txt", "a");
        fprintf(f2, "%.2f\n", timeToCalculate);
        fclose(f2);

        printf("time to calculate = %.2f milliseconds\n", timeToCalculate);

        //printf("determinant %f\n", determinant);

        //destroy events
        hipEventDestroy(begin);
        hipEventDestroy(end);
        //free memory
        hipFree(dev_data);
        free(data);
    }
    return 0;
}

//function definitions

__global__ void editRows(double *data, int matrixOrder, int rowNumber) {
    __syncthreads(); //wait for all threads
    unsigned int idx = blockIdx.x; //get index of block
    if (idx > rowNumber && idx <= matrixOrder - 1) {
        //get divider
        double divider = data[matrixOrder * idx + rowNumber] / data[rowNumber * matrixOrder + rowNumber];
        for (int j = rowNumber; j < matrixOrder; j++)
            //row edit
            data[matrixOrder * idx + j] -= data[rowNumber * matrixOrder + j] * divider;
    }
}

__host__ double gaussDeterminant(double *data, double *dev_data, int matrixOrder) {
    double determinant = 0;
    //matrix to triangle
    for (int i = 0; i < matrixOrder - 1; i++) {
        editRows<<<matrixOrder, 1>>>(dev_data, matrixOrder, i);
        hipDeviceSynchronize(); // synchronize device
    }
    hipDeviceSynchronize(); //synchronize threads
    //copy memory from device to CPU
    hipMemcpy(data, dev_data, matrixOrder * matrixOrder * sizeof(double), hipMemcpyDeviceToHost);
    //calculate determinant
    determinant = data[0];
    for (int i = 1; i < matrixOrder; i++) {
        determinant *= data[i * matrixOrder + i];
    }
    return determinant;
}
