
#include <hip/hip_runtime.h>
#include <iostream>

#include <cstdio>
#include <cstdlib>
#include <ctime>


__global__ void determinantGaussGPU(double* data, int matrixOrder, double* result);
__device__ void matrixToTriangle(double* data, int matrixOrder);

int main() {
    // FILE *f1, *f2;
    // f1 = fopen("Matrix Orders.txt", "w");
    // f2 = fopen("Time in milli seconds.txt", "w");
    int n = 0;

    int orders[1] = {3};

    for(n = 0; n < 1; n++){
        double *data = (double*) malloc(sizeof(double) * (orders[n] * orders[n]));

        srand(time(0));
        for (int i = 0; i < orders[n] * orders[n]; i++){
            data[i] =(double) (rand() % 11);
        }

        for (int i = 0; i != orders[n]; i++){
            for (int j = 0; j != orders[n]; j++)
                printf("%f ", data[i + orders[n] * j]);
            printf("\n");
        }

        double determinant = 1;

        printf("matrix order = %d\n", orders[n]);

        double *dev_det;
        double *dev_data;

        hipMalloc((void**)&dev_data, sizeof(double) * (orders[n] * orders[n]));
        hipMemcpy(dev_data, data, sizeof(double) * orders[n] * orders[n], hipMemcpyHostToDevice);

        hipMalloc(&dev_det, sizeof(double));
        hipMemcpy(dev_det, &determinant, sizeof(double), hipMemcpyHostToDevice);

        float timeToCalculate = 0.0;
        hipEvent_t begin, end;

        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, 0);

        determinantGaussGPU <<<orders[n], orders[n]>>>(dev_data, orders[n], dev_det);

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&timeToCalculate, begin, end);
        printf("time to calculate = %.2f milliseconds\n", timeToCalculate);


        hipMemcpy(&determinant, dev_det, sizeof(double), hipMemcpyDeviceToHost);

        printf("determinant %f", determinant);

        hipEventDestroy(begin);
        hipEventDestroy(end);

        hipFree(dev_det);
        hipFree(dev_data);
        free(data);
    }


    return 0;
}

__global__ void determinantGaussGPU(double* data, int matrixOrder, double* result){
    unsigned int row = blockIdx.x;

    matrixToTriangle(data, matrixOrder);

    *result *= data[row + blockDim.x * row];
}


__device__ void matrixToTriangle(double* data, int matrixOrder){
    unsigned int row = blockIdx.x;
    unsigned int col = threadIdx.x;

    double divider = 0.0;
    if (data[row + blockDim.x * row] != 0)
        divider = data[col + blockDim.x * row] / data[row + blockDim.x * row];
    for(int k = 0; k < matrixOrder; k++){
        data[col + blockDim.x * k] -= data[row + blockDim.x] * divider;
    }
}
