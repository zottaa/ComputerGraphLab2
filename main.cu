#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>

__global__ void editRows(double *data, int matrixOrder, int rowNumber);

__host__ double gaussDeterminant (double* data, double* dev_data, int matrixOrder);

int main() {
    FILE *f1, *f2;

    for (int n = 100; n < 3500; n += 20) {
        double *data = (double *) malloc(sizeof(double) * (n * n));

        srand(time(0));
        for (int i = 0; i < n * n; i++) {
            data[i] = (double) (rand() % 11);
        }

        double determinant = 1;

        f1 = fopen("Matrix Orders.txt", "a");
        printf("matrix order = %d\n", n);
        fprintf(f1, "%d\n", n);
        fclose(f1);

        double *dev_data;

        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, 0);

        hipMalloc((void **) &dev_data, sizeof(double) * (n * n));
        hipMemcpy(dev_data, data, sizeof(double) * n * n, hipMemcpyHostToDevice);

        float timeToCalculate = 0.0;

        determinant = gaussDeterminant(data, dev_data, n);

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        hipEventElapsedTime(&timeToCalculate, begin, end);

        f2 = fopen("Time in milli seconds GPU.txt", "a");
        fprintf(f2, "%.2f\n", timeToCalculate);
        fclose(f2);

        printf("time to calculate = %.2f milliseconds\n", timeToCalculate);

        //printf("determinant %f\n", determinant);

        hipEventDestroy(begin);
        hipEventDestroy(end);

        hipFree(dev_data);
        free(data);
    }
    return 0;
}

__global__ void editRows(double *data, int matrixOrder, int rowNumber) {
    __syncthreads();
    unsigned int idx = blockIdx.x;
    if (idx > rowNumber && idx <= matrixOrder - 1) {
        double divider = data[matrixOrder * idx + rowNumber] / data[rowNumber * matrixOrder + rowNumber];
        for (int j = rowNumber; j < matrixOrder; j++)
            data[matrixOrder * idx + j] -= data[rowNumber * matrixOrder + j] * divider;
    }
}

__host__ double gaussDeterminant(double *data, double *dev_data, int matrixOrder) {
    double determinant = 0;
    for (int i = 0; i < matrixOrder - 1; i++) {
        editRows<<<matrixOrder, 1>>>(dev_data, matrixOrder, i);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    hipMemcpy(data, dev_data, matrixOrder * matrixOrder * sizeof(double), hipMemcpyDeviceToHost);
    determinant = data[0];
    for (int i = 1; i < matrixOrder; i++) {
        determinant *= data[i * matrixOrder + i];
    }
    return determinant;
}
